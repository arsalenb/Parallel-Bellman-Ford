#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <limits.h>

#define INFINITY INT_MAX

#ifdef _WIN32
#include <windows.h>
long long gettime(void) {
    LARGE_INTEGER freq, counter;
    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&counter);
    return (counter.QuadPart * 1000000000LL) / freq.QuadPart;
}
#else
#include <time.h>
double gettime(void) {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}
#endif

typedef struct Edge {
    int u;
    int v;
    int w;
} Edge;

typedef struct Graph {
    int V;
    int E;
    struct Edge *edge;
} Graph;

// CUDA kernels
__global__ void init_kernel(int* d, int* p, int* mask, int* mask1, int V, int source) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < V) {
        d[idx] = (idx == source) ? 0 : INFINITY;
        p[idx] = -1;
        mask[idx] = (idx == source) ? 1 : 0;
        mask1[idx] = 0;
    }
}

__global__ void relax_kernel(Edge* edges, int* d, int* p, int* mask, int* mask1, int* flag, int E) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= E) return;

    Edge e = edges[idx];
    int u = e.u;
    int v = e.v;
    int w = e.w;

    if (mask[u] == 1) {
        if (d[u] != INFINITY) {
            int new_dist = d[u] + w;
            int old_dist = atomicMin(&d[v], new_dist);
            if (old_dist > new_dist) {
                p[v] = u;
                mask1[v] = 1;
                *flag = 1;
            }
        }
    }
}

__global__ void swap_kernel(int* mask, int* mask1, int* flag, int V) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < V) {
        mask[idx] = mask1[idx];
        mask1[idx] = 0;
    }
    if (idx == 0) {
        *flag = 0;
    }
}

Graph* createGraph(int V, int E) {
    Graph* graph = (Graph*)malloc(sizeof(Graph));
    graph->V = V;
    graph->E = E;
    graph->edge = (Edge*)malloc(graph->E * sizeof(Edge));
    return graph;
}

Graph* readGraph(char* filename) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        printf("Could not open file for reading\n");
        return NULL;
    }

    int V, E;
    fscanf(file, "%d\n", &E);
    fscanf(file, "%d\n", &V);
    Graph* graph = createGraph(V, E);
    int i = 0;
    char line[2048];

    while(fgets(line, sizeof(line), file)) {
        char* token = strtok(line, ":");
        int u = atoi(token);

        while((token = strtok(NULL, ";"))) {
            int v, w;
            sscanf(token, "%d,%d", &v, &w);
            if (i > 0 && u == graph->edge[i-1].u && v == graph->edge[i-1].v)
                continue;
            graph->edge[i].u = u;
            graph->edge[i].v = v;
            graph->edge[i].w = w;
            i++;
        }
    }
    fclose(file);
    return graph;
}

void display(int arr[], int size) {
    for (int i = 0; i < size; i++) {
        if (arr[i] == INFINITY) {
            printf("INF ");
        } else {
            printf("%d ", arr[i]);
        }
    }
    printf("\n");
}

void cuda_bellmanford(Graph* g, int source, bool debug) {
    Edge* d_edges;
    int *d_d, *d_p, *d_mask, *d_mask1, *d_flag;
    int *h_d, *h_p;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    hipMalloc(&d_edges, g->E * sizeof(Edge));
    hipMemcpy(d_edges, g->edge, g->E * sizeof(Edge), hipMemcpyHostToDevice);
    hipMalloc(&d_d, g->V * sizeof(int));
    hipMalloc(&d_p, g->V * sizeof(int));
    hipMalloc(&d_mask, g->V * sizeof(int));
    hipMalloc(&d_mask1, g->V * sizeof(int));
    hipMalloc(&d_flag, sizeof(int));

    // Record start time
    hipEventRecord(start, 0);

    // Initialize device arrays
    int threadsPerBlock = 512;
    int blocks = (g->V + threadsPerBlock - 1) / threadsPerBlock;
    init_kernel<<<blocks, threadsPerBlock>>>(d_d, d_p, d_mask, d_mask1, g->V, source);
    hipDeviceSynchronize();

    // Run relaxation until no more updates or V-1 iterations
    int relaxBlocks = (g->E + threadsPerBlock - 1) / threadsPerBlock;
    int h_flag;
    
    for (int i = 0; i < g->V - 1; i++) {
        h_flag = 0;
        hipMemcpy(d_flag, &h_flag, sizeof(int), hipMemcpyHostToDevice);
        
        relax_kernel<<<relaxBlocks, threadsPerBlock>>>(d_edges, d_d, d_p, d_mask, d_mask1, d_flag, g->E);
        hipDeviceSynchronize();
        
        hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
        if (h_flag == 0) {
            break;
        }
        
        swap_kernel<<<blocks, threadsPerBlock>>>(d_mask, d_mask1, d_flag, g->V);
        hipDeviceSynchronize();
    }

    // Record stop time and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float executionTime;
    hipEventElapsedTime(&executionTime, start, stop);

    // Allocate host memory for results and copy
    h_d = (int*)malloc(g->V * sizeof(int));
    h_p = (int*)malloc(g->V * sizeof(int));
    hipMemcpy(h_d, d_d, g->V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_p, d_p, g->V * sizeof(int), hipMemcpyDeviceToHost);

    if (debug) {
        printf("Distance array: ");
        display(h_d, g->V);
        printf("Predecessor array: ");
        display(h_p, g->V);
    }
    printf("Total algorithm time: %.3f milliseconds\n", executionTime);

    // Cleanup
    free(h_d);
    free(h_p);
    hipFree(d_edges);
    hipFree(d_d);
    hipFree(d_p);
    hipFree(d_mask);
    hipFree(d_mask1);
    hipFree(d_flag);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char *argv[]) {
	if (argc < 2 || argc > 3) {
        fprintf(stderr, "Usage: %s <graph_file_number> [debug]\n", argv[0]);
        fprintf(stderr, "  debug: 0 (silent) or 1 (verbose, default)\n");
        return 1;
    }
	
	// Set debug mode (default to true if not specified)
    bool debug = true;
    if (argc == 3) {
        debug = (atoi(argv[2]) != 0);
    }

    char filename[50];
    sprintf(filename, "graphs/graph_%d.txt", atoi(argv[1]));
    Graph* g = readGraph(filename);
    if (!g) return 1;

    cuda_bellmanford(g, 0, debug);

    free(g->edge);
    free(g);

    return 0;
}